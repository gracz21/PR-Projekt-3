#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""


__global__ void reduce0(int *g_idata, int *g_odata, int size){

   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
   sdata[tid] = 0;
   if(i<size)
     sdata[tid] = g_idata[i];
   __syncthreads();

	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}

   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

hipError_t reduceWithCuda(int *input, int size) {
	int output = 0;
	//Liczba w�tk�w na blok
	int threadsPerBlock = 1024;
	//Liczba blok�w (na pocz�tku)
	int totalBlocks = (size+(threadsPerBlock-1))/threadsPerBlock;

	//Wektor wej�ciowy i wyj�ciowy device
	int *dev_i, *dev_o;
	hipError_t cudaStatus;
	
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	
	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_i, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_o, totalBlocks * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_i, input, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	  
	bool turn = true;
	
	// Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    cudaStatus = hipEventCreate(&start);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event!\n");
        goto Error;
    }

    hipEvent_t stop;
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event!\n");
        goto Error;
    }

    cudaStatus = hipEventRecord(start, NULL);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event\n");
        goto Error;
    }
	  
	while(true) {	
		if(turn) {  
		  reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(dev_i, dev_o, size);
		  turn = false;
		} else {
		  reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(dev_o, dev_i, size);
		  turn = true;
		}
		
		if(totalBlocks == 1) break;
		
		size = totalBlocks;
		totalBlocks = ceil((double)totalBlocks/threadsPerBlock);
	}
	
    cudaStatus = hipEventRecord(stop, NULL);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event!\n");
        goto Error;
    }

    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event!\n");
        goto Error;
    }
	
	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduce0 launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
	
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }
	
	float msecTotal = 0.0f;
    cudaStatus = hipEventElapsedTime(&msecTotal, start, stop);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
	  
	if(turn) {
		cudaStatus = hipMemcpy(&output, &dev_i[0], sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	} else {
		cudaStatus = hipMemcpy(&output, &dev_o[0], sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}
	
	printf("Wynik to: %d, w czasie: %f\n", output, msecTotal);
	
Error:
    hipFree(dev_i);
    hipFree(dev_o);
	
	return cudaStatus;
}

int main(void) {
	//Deklaracja rozmiaru
	int size = 939289;
	//Wektor wej�ciowy hosta
	int *input = (int*)malloc(size * sizeof(int));
	for(int i = 0; i < size; i++)
		input[i] = 1;

	hipError_t cudaStatus = reduceWithCuda(input, size);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "reduceWithCuda failed!");
        return 1;
    }
	
	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
	free(input);

	return 0;
}